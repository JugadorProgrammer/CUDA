#include "hip/hip_runtime.h"
﻿#pragma once
#ifndef __INTELLISENSE_

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <cmath>

#define ll long long
#define MAX_TREAD_COUNT 1024
#define SHARED_MEMORY_SIZE MAX_TREAD_COUNT * sizeof(ll)

#define DELETE_IF_EXISTS(ptr) \
    if (ptr) \
    { \
        delete ptr; \
    }

#define DELETE_ARRAY_IF_EXISTS(ptr) \
    if (ptr) \
    { \
        delete[] ptr; \
    }

#define CHECK_CUDA_ERROR(cudaStatus, message) \
    if ((cudaStatus) != hipSuccess) \
    { \
        fprintf(stderr, message); \
		fprintf(stderr, "CUDA error string:  %s\n", hipGetErrorString(cudaStatus)); \
        goto Finish; \
    }

#define PRINT_CUDA_ERROR(cudaStatus, message) \
    if ((cudaStatus) != hipSuccess) \
    { \
        fprintf(stderr, message); \
		fprintf(stderr, "CUDA error string:  %s\n", hipGetErrorString(cudaStatus)); \
    }

__host__ void fillArray(ll* arr, const size_t size)
{
	for (size_t i = 0; i < size; ++i)
	{
		arr[i] = i + 1;
	}
}

__host__ void printArray(ll* arr, const size_t size)
{
	printf("[ ");
	for (size_t i = 0; i < size; ++i)
	{
		printf("%lld ", arr[i]);
	}
	printf("]\n");
}

__host__ void printDeviceProperties(const hipDeviceProp_t& deviceProp)
{
	// Основная информация
	printf("\n\nGPU: %s\n", deviceProp.name);
	printf("Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
	printf("Global Memory: %.2f GB\n", deviceProp.totalGlobalMem / 1073741824.0);

	// Блоки и сетка
	printf("Max Threads per Block: %d\n", deviceProp.maxThreadsPerBlock);
	printf("Max Block Dim: (%d, %d, %d)\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
	printf("Max Grid Dim: (%d, %d, %d)\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);

	// Аппаратные характеристики
	printf("Max blocks per multiprocessor: %d\n", deviceProp.maxBlocksPerMultiProcessor);
	printf("Multiprocessors: %d\n", deviceProp.multiProcessorCount);
	printf("Clock Rate: %.2f GHz\n", deviceProp.clockRate * 1e-6f);
	printf("Shared Memory per Block: %zu KB\n", deviceProp.sharedMemPerBlock / 1024);

	// Дополнительно
	printf("Warp Size: %d\n", deviceProp.warpSize);
	printf("Concurrent Kernels: %s\n", deviceProp.concurrentKernels ? "Yes" : "No");
	printf("Integrated GPU: %s\n\n\n", deviceProp.integrated ? "Yes" : "No");
}

// Фаза Up-sweep (редукция)
__global__ void upsweep_kernel(ll* arr, int size, int stride)
{
	int threadId = blockDim.x * blockIdx.x + threadIdx.x;
	// Должны работать потоки с индексами: stride*2-1, stride*4-1, stride*6-1, ...
	if (threadId >= size)
	{
		return;
	}

	// Правильное условие: поток должен быть на позиции (k * 2 * stride - 1)
	if ((threadId + 1) % (2 * stride) == 0)
	{
		int left_idx = threadId - stride;
		if (left_idx >= 0)
		{
			arr[threadId] += arr[left_idx];
		}
	}
}

// Фаза Down-sweep (распространение)
__global__ void downsweep_kernel(ll* arr, size_t size, int stride)
{
	int threadId = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadId >= size)
	{
		return;
	}

	// Потоки должны быть на позициях: stride*2-1, stride*4-1, stride*6-1, ...
	if ((threadId + 1) % (2 * stride) == 0)
	{
		int left_idx = threadId - stride;
		if (left_idx >= 0) 
		{
			// Сохраняем значение левого элемента
			ll temp = arr[left_idx];
			// Перемещаем текущее значение в левый элемент
			arr[left_idx] = arr[threadId];
			// Добавляем сохраненное значение к текущему
			arr[threadId] += temp;
		}
	}
}

__host__ hipError_t prefixAmount(ll* source, int size)
{
	hipError_t cudaStatus;
	for (int stride = 1; stride < size; stride *= 2)
	{
		int blocks_per_grid = ceill((double)size / MAX_TREAD_COUNT);
		if (blocks_per_grid == 0)
		{
			blocks_per_grid = 1;
		}

		upsweep_kernel<<<blocks_per_grid, MAX_TREAD_COUNT>>>(source, size, stride);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess)
		{
			return cudaStatus;
		}
	}

	ll total_sum;
	// Сохраняем общую сумму и обнуляем последний элемент
	hipMemcpy(&total_sum, &source[size - 1], sizeof(ll), hipMemcpyDeviceToHost);
	hipMemset(&source[size - 1], 0, sizeof(ll));

	for (int stride = size / 2; stride >= 1; stride /= 2)
	{
		int blocks_per_grid = ceill((double)size / MAX_TREAD_COUNT);
		if (blocks_per_grid == 0)
		{
			blocks_per_grid = 1;
		}

		downsweep_kernel << <blocks_per_grid, MAX_TREAD_COUNT >> > (source, size, stride);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess)
		{
			return cudaStatus;
		}
	}
	return hipSuccess;
}

int main()
{
	srand(time(NULL));

	ll* source = NULL, * devSource = NULL, * result = NULL, * devResult = NULL;
	hipError_t cudaStatus;
	hipEvent_t start, stop;
	hipDeviceProp_t deviceProp;
	float milliseconds = 0;

	const size_t arraySize = 8;
	const dim3 blockDim(MAX_TREAD_COUNT), gridDim((size_t)ceil(arraySize / ((double)blockDim.x)));

	source = new ll[arraySize];
	fillArray(source, arraySize);
	printArray(source, arraySize);

	///////////////////////////////////////GPU/////////////////////////////////////////////////////
	cudaStatus = hipEventCreate(&start);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventCreate(&start) failed!");

	cudaStatus = hipEventCreate(&stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventCreate(&stop) failed!");

	cudaStatus = hipGetDeviceProperties(&deviceProp, 0);
	CHECK_CUDA_ERROR(cudaStatus, "hipGetDeviceProperties failed!");

	printDeviceProperties(deviceProp);

	cudaStatus = hipMalloc(&devSource, arraySize * sizeof(ll));
	CHECK_CUDA_ERROR(cudaStatus, "hipMalloc(&devSource failed!");

	cudaStatus = hipMalloc(&devResult, arraySize * sizeof(ll));
	CHECK_CUDA_ERROR(cudaStatus, "hipMalloc(&devResult failed!");

	cudaStatus = hipMemcpy(devSource, source, arraySize * sizeof(ll), hipMemcpyHostToDevice);
	CHECK_CUDA_ERROR(cudaStatus, "hipMemcpy(devSource failed!");

	printf("GPU start calculation\n");
	cudaStatus = hipEventRecord(start);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventRecord(&start) failed!");

	cudaStatus = prefixAmount(devSource, arraySize);
	CHECK_CUDA_ERROR(cudaStatus, "prefixAmount failed!");

	cudaStatus = hipGetLastError();
	CHECK_CUDA_ERROR(cudaStatus, "hipGetLastError failed!");

	cudaStatus = hipDeviceSynchronize();
	CHECK_CUDA_ERROR(cudaStatus, "hipDeviceSynchronize failed!");

	cudaStatus = hipEventRecord(stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventRecord(&stop) failed!");

	// Ждем завершения всех операций
	cudaStatus = hipEventSynchronize(stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventSynchronize(&stop) failed!");

	cudaStatus = hipEventElapsedTime(&milliseconds, start, stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventElapsedTime failed!");

	result = new ll[arraySize];
	cudaStatus = hipMemcpy(result, devSource, arraySize * sizeof(ll), hipMemcpyDeviceToHost);
	CHECK_CUDA_ERROR(cudaStatus, "hipMemcpy(&result failed!");

	printArray(result, arraySize);
	printf("GPU time: %f ms\n", milliseconds);

Finish:
	DELETE_ARRAY_IF_EXISTS(source);
	DELETE_ARRAY_IF_EXISTS(result);

	// Освобождаем ресурсы
	cudaStatus = hipEventDestroy(start);
	PRINT_CUDA_ERROR(cudaStatus, "hipEventDestroy(start failed!");

	cudaStatus = hipEventDestroy(stop);
	PRINT_CUDA_ERROR(cudaStatus, "hipEventDestroy(stop failed!");

	if (devSource)
	{
		cudaStatus = hipFree(devSource);
		PRINT_CUDA_ERROR(cudaStatus, "hipFree(devSource failed!");
	}

	if (devResult)
	{
		cudaStatus = hipFree(devResult);
		PRINT_CUDA_ERROR(cudaStatus, "hipFree(devResult failed!");
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	PRINT_CUDA_ERROR(cudaStatus, "hipDeviceReset failed!");

	system("pause");
	return 0;
}
#endif
