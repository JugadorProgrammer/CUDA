#include "hip/hip_runtime.h"
﻿#pragma once
#ifndef __INTELLISENSE_

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <cmath>
#include <opencv2/opencv.hpp>

#define CLAMP(x, min, max) ((x) < (min) ? (min) : ((x) > (max) ? (max) : (x)))
#define ll long long
#define TILE_SIZE 32
#define CALC_TIME_MS(start, end) (((double)((end) - (start)) * 1000.0) / CLOCKS_PER_SEC)

#define DELETE_IF_EXISTS(ptr) \
    if (ptr) \
    { \
        delete ptr; \
    }

#define DELETE_ARRAY_IF_EXISTS(ptr) \
    if (ptr) \
    { \
        delete[] ptr; \
    }

#define CHECK_CUDA_ERROR(cudaStatus, message) \
    if ((cudaStatus) != hipSuccess) \
    { \
        fprintf(stderr, message); \
		fprintf(stderr, "CUDA error string:  %s\n", hipGetErrorString(cudaStatus)); \
        goto Finish; \
    }

#define PRINT_CUDA_ERROR(cudaStatus, message) \
    if ((cudaStatus) != hipSuccess) \
    { \
        fprintf(stderr, message); \
		fprintf(stderr, "CUDA error string:  %s\n", hipGetErrorString(cudaStatus)); \
    }

__host__ void printDeviceProperties(const hipDeviceProp_t& deviceProp)
{
	// Основная информация
	printf("\n\nGPU: %s\n", deviceProp.name);
	printf("Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
	printf("Global Memory: %.2f GB\n", deviceProp.totalGlobalMem / 1073741824.0);

	// Блоки и сетка
	printf("Max Threads per Block: %d\n", deviceProp.maxThreadsPerBlock);
	printf("Max Block Dim: (%d, %d, %d)\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
	printf("Max Grid Dim: (%d, %d, %d)\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);

	// Аппаратные характеристики
	printf("Max blocks per multiprocessor: %d\n", deviceProp.maxBlocksPerMultiProcessor);
	printf("Multiprocessors: %d\n", deviceProp.multiProcessorCount);
	printf("Clock Rate: %.2f GHz\n", deviceProp.clockRate * 1e-6f);
	printf("Shared Memory per Block: %zu KB\n", deviceProp.sharedMemPerBlock / 1024);

	// Дополнительно
	printf("Warp Size: %d\n", deviceProp.warpSize);
	printf("Concurrent Kernels: %s\n", deviceProp.concurrentKernels ? "Yes" : "No");
	printf("Integrated GPU: %s\n\n\n", deviceProp.integrated ? "Yes" : "No");
}

__global__ void gaussianBlurKernelShared(uchar* image, uchar* output, int width, int height, float* kernel, size_t kernelSize)
{
	int tileStartX = blockIdx.x * blockDim.x;
	int tileStartY = blockIdx.y * blockDim.y;
	int tileEndX = (blockIdx.x + 1) * blockDim.x;
	int tileEndY = (blockIdx.y + 1) * blockDim.y;

	int x = tileStartX + threadIdx.x;
	int y = tileStartY + threadIdx.y;

	// Проверка выхода за границы изображения
	if (x < 2 || y < 2 || x > width - 2 || y > height - 2)
	{
		return;
	}

	int radius = kernelSize / 2;
	float sumB = 0.0f, sumG = 0.0f, sumR = 0.0f, sumA = 0.0f;
	__shared__ uchar tile[TILE_SIZE * TILE_SIZE * 4];

	tile[(y % TILE_SIZE * TILE_SIZE + x % TILE_SIZE) * 4] = image[(y * width + x) * 4];
	tile[(y % TILE_SIZE * TILE_SIZE + x % TILE_SIZE) * 4 + 1] = image[(y * width + x) * 4 + 1];
	tile[(y % TILE_SIZE * TILE_SIZE + x % TILE_SIZE) * 4 + 2] = image[(y * width + x) * 4 + 2];
	tile[(y % TILE_SIZE * TILE_SIZE + x % TILE_SIZE) * 4 + 3] = image[(y * width + x) * 4 + 3];
	__syncthreads();
	// Применяем ядро Гаусса к каждому каналу BGRA
	for (int ky = -radius; ky <= radius; ++ky)
	{
		for (int kx = -radius; kx <= radius; ++kx)
		{
			int posX = x + kx;
			int posY = y + ky;

			// Безопасная обработка граничных условий (clamp to edge)
			if (posX > width - 1)
			{
				posX = width - 1;
			}
			if (posY > height - 1)
			{
				posY = height - 1;
			}

			int kernelIndex = (ky + radius) * kernelSize + (kx + radius);
			float kernelValue = kernel[kernelIndex];
			if (posX < tileStartX || posX >= tileEndX || posY < tileStartY || posY >= tileEndY)
			{
				int pixelIndex = (posY * width + posX) * 4;
				//// Умножаем каждый канал на коэффициент ядра
				sumB += image[pixelIndex] * kernelValue;     // Blue
				sumG += image[pixelIndex + 1] * kernelValue; // Green
				sumR += image[pixelIndex + 2] * kernelValue; // Red
				sumA += image[pixelIndex + 3] * kernelValue; // Alpha
				continue;
			}

			int pixelIndex = ((posY % TILE_SIZE) * TILE_SIZE + posX % TILE_SIZE) * 4;
			//// Умножаем каждый канал на коэффициент ядра
			sumB += tile[pixelIndex] * kernelValue;     // Blue
			sumG += tile[pixelIndex + 1] * kernelValue; // Green
			sumR += tile[pixelIndex + 2] * kernelValue; // Red
			sumA += tile[pixelIndex + 3] * kernelValue; // Alpha
		}
	}

	int outputIndex = (y * width + x) * 4;
	output[outputIndex] = (uchar)(sumB);     // Blue
	output[outputIndex + 1] = (uchar)(sumG); // Green
	output[outputIndex + 2] = (uchar)(sumR); // Red
	output[outputIndex + 3] = (uchar)(sumA); // Alpha
}

__global__ void gaussianBlurKernel(uchar* image, uchar* output, int width, int height, float* kernel, size_t kernelSize)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x);
	int y = (blockIdx.y * blockDim.y + threadIdx.y);

	// Проверка выхода за границы изображения
	if (x < 2 || y < 2 || x > width - 2 || y > height - 2)
	{
		return;
	}

	int radius = kernelSize / 2;
	float sumB = 0.0f, sumG = 0.0f, sumR = 0.0f, sumA = 0.0f;

	// Применяем ядро Гаусса к каждому каналу BGRA
	for (int ky = -radius; ky <= radius; ++ky)
	{
		for (int kx = -radius; kx <= radius; ++kx)
		{
			int posX = x + kx;
			int posY = y + ky;

			// Безопасная обработка граничных условий (clamp to edge)
			if (posX > width - 1)
			{
				posX = width - 1;
			}
			if (posY > height - 1)
			{
				posY = height - 1;
			}

			// Получаем индекс пикселя в BGRA формате (4 канала на пиксель)
			int pixelIndex = (posY * width + posX) * 4;
			int kernelIndex = (ky + radius) * kernelSize + (kx + radius);
			float kernelValue = kernel[kernelIndex];

			//// Умножаем каждый канал на коэффициент ядра
			sumB += image[pixelIndex] * kernelValue;     // Blue
			sumG += image[pixelIndex + 1] * kernelValue; // Green
			sumR += image[pixelIndex + 2] * kernelValue; // Red
			sumA += image[pixelIndex + 3] * kernelValue; // Alpha
		}
	}

	int outputIndex = (y * width + x) * 4;
	output[outputIndex] = (uchar)(sumB);     // Blue
	output[outputIndex + 1] = (uchar)(sumG); // Green
	output[outputIndex + 2] = (uchar)(sumR); // Red
	output[outputIndex + 3] = (uchar)(sumA); // Alpha
}

__host__ void gaussianBlurKernelCPU(uchar* image, uchar* output, int width, int height, const float* kernel, const size_t kernelSize)
{
	const int radius = kernelSize / 2;
	// Создаем временный буфер
	uchar* temp = new uchar[width * height];
	memcpy(temp, image, width * height);

	// Применяем фильтр по горизонтали
	for (int y = 0; y < height; ++y) 
	{
		for (int x = 0; x < width; ++x) 
		{
			if (x < 2 || y < 2 || x > width - 2 || y > height - 2)
			{
				continue;
			}

			float sumB = 0.0f, sumG = 0.0f, sumR = 0.0f, sumA = 0.0f;

			// Применяем ядро Гаусса к каждому каналу BGRA
			for (int ky = -radius; ky <= radius; ++ky)
			{
				for (int kx = -radius; kx <= radius; ++kx)
				{
					int posX = x + kx;
					int posY = y + ky;

					// Безопасная обработка граничных условий (clamp to edge)
					if (posX > width - 1)
					{
						posX = width - 1;
					}
					if (posY > height - 1)
					{
						posY = height - 1;
					}

					// Получаем индекс пикселя в BGRA формате (4 канала на пиксель)
					int pixelIndex = (posY * width + posX) * 4;
					int kernelIndex = (ky + radius) * kernelSize + (kx + radius);
					float kernelValue = kernel[kernelIndex];

					//// Умножаем каждый канал на коэффициент ядра
					sumB += image[pixelIndex] * kernelValue;     // Blue
					sumG += image[pixelIndex + 1] * kernelValue; // Green
					sumR += image[pixelIndex + 2] * kernelValue; // Red
					sumA += image[pixelIndex + 3] * kernelValue; // Alpha
				}
			}

			int outputIndex = (y * width + x) * 4;
			output[outputIndex] = (uchar)(sumB);     // Blue
			output[outputIndex + 1] = (uchar)(sumG); // Green
			output[outputIndex + 2] = (uchar)(sumR); // Red
			output[outputIndex + 3] = (uchar)(sumA); // Alpha
		}
	}
}

__host__ void CPU(const cv::Mat& image, const float* kernel, const size_t kernelSize)
{
	cv::Mat resultMat = cv::Mat(image.rows, image.cols, image.type());

	printf("CPU start calculation\n");
	clock_t start, end;

	start = clock();
	gaussianBlurKernelCPU(image.data, resultMat.data, image.cols, image.rows, kernel, kernelSize);
	end = clock();

	float milliseconds = CALC_TIME_MS(start, end);
	printf("CPU: Time = %f ms\n", milliseconds);
	cv::imwrite("СPU_Result.png", resultMat);
}

__host__ void GPU(const cv::Mat& image, const float* kernel, const size_t kernelSize)
{
	hipError_t cudaStatus;
	hipEvent_t start, stop;
	int width = image.cols, height = image.rows, channelsCount = image.channels();

	float* devKernel = NULL, milliseconds = 0;
	cv::Mat resultMat;
	uchar* devImage = NULL, * devResultImage = NULL;
	const dim3 blockDim(TILE_SIZE, TILE_SIZE),
		gridDim((size_t)std::ceil((double)width / (double)TILE_SIZE), (size_t)std::ceil((double)height / (double)TILE_SIZE));

	///////////////////////////////////////GPU/////////////////////////////////////////////////////
	cudaStatus = hipEventCreate(&start);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventCreate(&start) failed!");

	cudaStatus = hipEventCreate(&stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventCreate(&stop) failed!");

	cudaStatus = hipMalloc(&devImage, width * height * channelsCount * sizeof(uchar));
	CHECK_CUDA_ERROR(cudaStatus, "hipMalloc(&devImage failed!");

	cudaStatus = hipMalloc(&devResultImage, width * height * channelsCount * sizeof(uchar));
	CHECK_CUDA_ERROR(cudaStatus, "hipMalloc(&devResultImage failed!");

	cudaStatus = hipMalloc(&devKernel, kernelSize * kernelSize * sizeof(float));
	CHECK_CUDA_ERROR(cudaStatus, "hipMalloc(&devKernel failed!");

	cudaStatus = hipMemcpy(devKernel, kernel, kernelSize * kernelSize * sizeof(float), hipMemcpyHostToDevice);
	CHECK_CUDA_ERROR(cudaStatus, "hipMemcpy(devKernel failed!");

	cudaStatus = hipMemcpy(devImage, image.data, width * height * channelsCount * sizeof(uchar), hipMemcpyHostToDevice);
	CHECK_CUDA_ERROR(cudaStatus, "hipMemcpy(devImage failed!");

	printf("GPU start calculation\n");
	cudaStatus = hipEventRecord(start);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventRecord(&start) failed!");

	gaussianBlurKernel << <gridDim, blockDim >> > (devImage, devResultImage, width, height, devKernel, kernelSize);

	cudaStatus = hipGetLastError();
	CHECK_CUDA_ERROR(cudaStatus, "hipGetLastError failed!");

	cudaStatus = hipDeviceSynchronize();
	CHECK_CUDA_ERROR(cudaStatus, "hipDeviceSynchronize failed!");

	cudaStatus = hipEventRecord(stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventRecord(&stop) failed!");

	// Ждем завершения всех операций
	cudaStatus = hipEventSynchronize(stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventSynchronize(&stop) failed!");

	cudaStatus = hipEventElapsedTime(&milliseconds, start, stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventElapsedTime failed!");

	resultMat = cv::Mat(image.rows, image.cols, image.type());
	cudaStatus = hipMemcpy(resultMat.data, devResultImage, width * height * channelsCount * sizeof(uchar), hipMemcpyDeviceToHost);
	CHECK_CUDA_ERROR(cudaStatus, "hipMemcpy(devResultImage failed!");

	printf("GPU time: %f ms\n", milliseconds);

	cv::imwrite("GPU_Result.png", resultMat);
Finish:
	resultMat.release();

	// Освобождаем ресурсы
	cudaStatus = hipEventDestroy(start);
	PRINT_CUDA_ERROR(cudaStatus, "hipEventDestroy(start failed!");

	cudaStatus = hipEventDestroy(stop);
	PRINT_CUDA_ERROR(cudaStatus, "hipEventDestroy(stop failed!");

	if (devImage)
	{
		cudaStatus = hipFree(devImage);
		PRINT_CUDA_ERROR(cudaStatus, "hipFree(devImage failed!");
	}

	if (devResultImage)
	{
		cudaStatus = hipFree(devResultImage);
		PRINT_CUDA_ERROR(cudaStatus, "hipFree(devResultImage failed!");
	}

	if (devKernel)
	{
		cudaStatus = hipFree(devKernel);
		PRINT_CUDA_ERROR(cudaStatus, "hipFree(devKernel failed!");
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	PRINT_CUDA_ERROR(cudaStatus, "hipDeviceReset failed!");
}

__host__ void GPUShared(const cv::Mat& image, const float* kernel, const size_t kernelSize)
{
	hipError_t cudaStatus;
	hipEvent_t start, stop;
	int width = image.cols, height = image.rows, channelsCount = image.channels();

	float* devKernel = NULL, milliseconds = 0;
	cv::Mat resultMat;
	uchar* devImage = NULL, * devResultImage = NULL;
	const dim3 blockDim(TILE_SIZE, TILE_SIZE),
		gridDim((size_t)std::ceil((double)width / (double)TILE_SIZE), (size_t)std::ceil((double)height / (double)TILE_SIZE));

	///////////////////////////////////////GPU/////////////////////////////////////////////////////
	cudaStatus = hipEventCreate(&start);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventCreate(&start) failed!");

	cudaStatus = hipEventCreate(&stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventCreate(&stop) failed!");

	cudaStatus = hipMalloc(&devImage, width * height * channelsCount * sizeof(uchar));
	CHECK_CUDA_ERROR(cudaStatus, "hipMalloc(&devImage failed!");

	cudaStatus = hipMalloc(&devResultImage, width * height * channelsCount * sizeof(uchar));
	CHECK_CUDA_ERROR(cudaStatus, "hipMalloc(&devResultImage failed!");

	cudaStatus = hipMalloc(&devKernel, kernelSize * kernelSize * sizeof(float));
	CHECK_CUDA_ERROR(cudaStatus, "hipMalloc(&devKernel failed!");

	cudaStatus = hipMemcpy(devKernel, kernel, kernelSize * kernelSize * sizeof(float), hipMemcpyHostToDevice);
	CHECK_CUDA_ERROR(cudaStatus, "hipMemcpy(devKernel failed!");

	cudaStatus = hipMemcpy(devImage, image.data, width * height * channelsCount * sizeof(uchar), hipMemcpyHostToDevice);
	CHECK_CUDA_ERROR(cudaStatus, "hipMemcpy(devImage failed!");

	printf("Shared GPU start calculation\n");
	cudaStatus = hipEventRecord(start);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventRecord(&start) failed!");

	gaussianBlurKernelShared << <gridDim, blockDim >> > (devImage, devResultImage, width, height, devKernel, kernelSize);

	cudaStatus = hipGetLastError();
	CHECK_CUDA_ERROR(cudaStatus, "hipGetLastError failed!");

	cudaStatus = hipDeviceSynchronize();
	CHECK_CUDA_ERROR(cudaStatus, "hipDeviceSynchronize failed!");

	cudaStatus = hipEventRecord(stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventRecord(&stop) failed!");

	// Ждем завершения всех операций
	cudaStatus = hipEventSynchronize(stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventSynchronize(&stop) failed!");

	cudaStatus = hipEventElapsedTime(&milliseconds, start, stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventElapsedTime failed!");

	resultMat = cv::Mat(image.rows, image.cols, image.type());
	cudaStatus = hipMemcpy(resultMat.data, devResultImage, width * height * channelsCount * sizeof(uchar), hipMemcpyDeviceToHost);
	CHECK_CUDA_ERROR(cudaStatus, "hipMemcpy(devResultImage failed!");

	printf("GPU Shared time: %f ms\n", milliseconds);

	cv::imwrite("Shared_Result.png", resultMat);
Finish:
	resultMat.release();

	// Освобождаем ресурсы
	cudaStatus = hipEventDestroy(start);
	PRINT_CUDA_ERROR(cudaStatus, "hipEventDestroy(start failed!");

	cudaStatus = hipEventDestroy(stop);
	PRINT_CUDA_ERROR(cudaStatus, "hipEventDestroy(stop failed!");

	if (devImage)
	{
		cudaStatus = hipFree(devImage);
		PRINT_CUDA_ERROR(cudaStatus, "hipFree(devImage failed!");
	}

	if (devResultImage)
	{
		cudaStatus = hipFree(devResultImage);
		PRINT_CUDA_ERROR(cudaStatus, "hipFree(devResultImage failed!");
	}

	if (devKernel)
	{
		cudaStatus = hipFree(devKernel);
		PRINT_CUDA_ERROR(cudaStatus, "hipFree(devKernel failed!");
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	PRINT_CUDA_ERROR(cudaStatus, "hipDeviceReset failed!");
}

__host__ float* createKernel(size_t kernelSize, float sigma = 1.0)
{
	kernelSize |= 1; // должен быть нечётным
	float* kernel = new float[kernelSize * kernelSize], sum = 0;
	int radius = kernelSize / 2;

	for (int y = -radius; y <= radius; ++y)
	{
		for (int x = -radius; x <= radius; ++x)
		{
			float value = exp(-(x * x + y * y) / (2 * sigma * sigma));
			kernel[(y + radius) * kernelSize + (x + radius)] = value;
			sum += value;
		}
	}

	// Нормализация
	for (int i = 0; i < kernelSize * kernelSize; ++i)
	{
		kernel[i] /= sum;
	}

	return kernel;
}

__host__ cv::Mat converToBgra(const char* fileName)
{
	cv::Mat imageBGRA, source = cv::imread(fileName, cv::IMREAD_UNCHANGED);
	cv::cvtColor(source, imageBGRA, cv::COLOR_BGR2BGRA);
	source.release();
	return imageBGRA;
}

long main()
{
	const size_t kernelSize = 5;
	cv::Mat image = converToBgra("source.png");
	float* kernel = createKernel(kernelSize);
	hipDeviceProp_t deviceProp;

	hipGetDeviceProperties(&deviceProp, 0);
	printDeviceProperties(deviceProp);

	CPU(image, kernel, kernelSize);
	GPU(image, kernel, kernelSize);
	GPUShared(image, kernel, kernelSize);

	delete[] kernel;
	image.release();
	system("pause");
	return 0;
}
#endif
