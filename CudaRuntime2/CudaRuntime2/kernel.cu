#include "hip/hip_runtime.h"
﻿#pragma once
#ifndef __INTELLISENSE_

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <cmath>

// размер блока или размер подматрицы
#define ll long long
#define FINAL_MATRIX_HEIGHT 200.0
#define FINAL_MATRIX_WIDTH 300.0
#define MAX_2D_TREAD_COUNT 32.0
#define TILE_SIZE (size_t)MAX_2D_TREAD_COUNT
#define CALC_TIME_MS(start, end) (((double)((end) - (start)) * 1000.0) / CLOCKS_PER_SEC)

#define DELETE_IF_EXISTS(ptr) \
    if (ptr) \
    { \
        delete ptr; \
    }

#define DELETE_ARRAY_IF_EXISTS(ptr) \
    if (ptr) \
    { \
        delete[] ptr; \
    }

#define CHECK_CUDA_ERROR(cudaStatus, message) \
    if ((cudaStatus) != hipSuccess) \
    { \
        fprintf(stderr, message); \
		fprintf(stderr, "CUDA error string:  %s\n", hipGetErrorString(cudaStatus)); \
        goto Finish; \
    }

#define PRINT_CUDA_ERROR(cudaStatus, message) \
    if ((cudaStatus) != hipSuccess) \
    { \
        fprintf(stderr, message); \
		fprintf(stderr, "CUDA error string:  %s\n", hipGetErrorString(cudaStatus)); \
    }

struct Size
{
	size_t width = 0;
	size_t height = 0;
};

__host__ void fillMatrix(ll* matrix, const struct Size size)
{
	for (size_t i = 0; i < size.width * size.height; ++i)
	{
		matrix[i] = i;
	}
}

__host__ void printMatrix(ll* matrix, const struct Size size, const char* matrixName)
{
	printf("Matrix %s:\n", matrixName);
	for (size_t i = 0; i < size.height; ++i)
	{
		for (size_t j = 0; j < size.width; ++j)
		{
			printf("%lld ", matrix[size.width * i + j]);
		}
		printf("\n");
	}
}

__host__ struct Size matrixMult(const ll* a, const ll* b, ll* result, struct Size aSize, struct Size bSize)
{
	struct Size resultSize;
	resultSize.width = bSize.width;
	resultSize.height = aSize.height;

	size_t n = aSize.width;
	for (size_t i = 0; i < resultSize.height; ++i)
	{
		for (size_t j = 0; j < resultSize.width; ++j)
		{
			size_t index = i * resultSize.width + j;
			result[index] = 0;
			for (size_t k = 0; k < n; ++k)
			{
				result[index] += a[i * aSize.width + k] * b[k * bSize.width + j];
			}
		}
	}

	return resultSize;
}

__host__ void printDeviceProperties(const hipDeviceProp_t& deviceProp)
{
	// Основная информация
	printf("\n\nGPU: %s\n", deviceProp.name);
	printf("Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
	printf("Global Memory: %.2f GB\n", deviceProp.totalGlobalMem / 1073741824.0);

	// Блоки и сетка
	printf("Max Threads per Block: %d\n", deviceProp.maxThreadsPerBlock);
	printf("Max Block Dim: (%d, %d, %d)\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
	printf("Max Grid Dim: (%d, %d, %d)\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);

	// Аппаратные характеристики
	printf("Max blocks per multiprocessor: %d\n", deviceProp.maxBlocksPerMultiProcessor);
	printf("Multiprocessors: %d\n", deviceProp.multiProcessorCount);
	printf("Clock Rate: %.2f GHz\n", deviceProp.clockRate * 1e-6f);
	printf("Shared Memory per Block: %zu KB\n", deviceProp.sharedMemPerBlock / 1024);

	// Дополнительно
	printf("Warp Size: %d\n", deviceProp.warpSize);
	printf("Concurrent Kernels: %s\n", deviceProp.concurrentKernels ? "Yes" : "No");
	printf("Integrated GPU: %s\n\n\n", deviceProp.integrated ? "Yes" : "No");
}

__global__ void matrixMultGPU(const ll* a, const ll* b, ll* result, struct Size* resultSize, struct Size aSize, struct Size bSize)
{
	size_t indexX = blockDim.x * blockIdx.x + threadIdx.x;
	size_t indexY = blockDim.y * blockIdx.y + threadIdx.y;

	if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0)
	{
		resultSize->width = bSize.width;
		resultSize->height = aSize.height;
	}

	if (indexX >= bSize.width || indexY >= aSize.height)
	{
		return;
	}

	ll sum = 0ll;
	for (size_t k = 0; k < aSize.width; ++k)
	{
		sum += a[indexY * aSize.width + k] * b[k * bSize.width + indexX];
	}

	size_t index = bSize.width * indexY + indexX;
	result[index] = sum;
}

__global__ void matrixMultGPUShared(const ll* a, const ll* b, ll* result, struct Size* resultSize, struct Size aSize, struct Size bSize)
{
	size_t indexX = blockDim.x * blockIdx.x + threadIdx.x;
	size_t indexY = blockDim.y * blockIdx.y + threadIdx.y;

	if (indexX >= bSize.width || indexY >= aSize.height)
	{
		return;
	}

	// индекс начала первой подматрицы А, которую
	// обрабатывает блок
	size_t aBegin = aSize.width * blockDim.y * blockIdx.y;
	// индекс конца подматрицы А, которую обрабатывает блок
	size_t aEnd = aBegin + aSize.width - 1;
	// шаг для перебора подматриц А
	size_t aStep = blockDim.x;
	// индекс начала первой подматрицы В, которую
	// обрабатывает блок
	size_t bBegin = blockDim.x * blockIdx.x;
	// шаг для перебора подматриц В
	size_t bStep = blockDim.y * bSize.width;

	// Выделение разделяемой памяти для подматриц
	__shared__ ll as[TILE_SIZE][TILE_SIZE];
	__shared__ ll bs[TILE_SIZE][TILE_SIZE];
	// переменная для вычисления элемента подматрицы
	ll sum = 0;
	for (size_t ia = aBegin, ib = bBegin; ia < aEnd; ia += aStep, ib += bStep)
	{
		// загрузка подматриц А и В из глобальной памяти в
		// разделяемую
		as[threadIdx.y][threadIdx.x] = a[ia + aSize.width * threadIdx.y + threadIdx.x];
		bs[threadIdx.y][threadIdx.x] = b[ib + bSize.width * threadIdx.y + threadIdx.x];
		// синхронизация нитей
		__syncthreads();
		// перемножение двух матриц
		for (size_t k = 0; k < blockDim.x; k++)
		{
			sum += as[threadIdx.y][k] * bs[k][threadIdx.x];
		}
		// синхронизация нитей
		__syncthreads();
	}

	size_t index = bSize.width * indexY + indexX;
	result[index] = sum;

	if (blockIdx.x == 0 && threadIdx.x == 0)
	{
		resultSize->width = bSize.width;
		resultSize->height = aSize.height;
	}
}

//__global__ void matrixMult(const double* A, const double* B, double* C, int Acols, int Bcols)
//{
//	// индекс начала первой подматрицы А, которую
//	// обрабатывает блок
//	int aBegin = Acols * blockDim.y * blockIdx.y;
//	// индекс конца подматрицы А, которую обрабатывает блок
//	int aEnd = aBegin + Acols - 1;
//	// шаг для перебора подматриц А
//	int aStep = blockDim.x;
//	// индекс начала первой подматрицы В, которую
//	// обрабатывает блок
//	int bBegin = blockDim.x * blockIdx.x;
//	// шаг для перебора подматриц В
//	int bStep = blockDim.y * Bcols;
//
//	// Выделение разделяемой памяти для подматриц
//	__shared__ double as[TILE_SIZE][TILE_SIZE];
//	__shared__ double bs[TILE_SIZE][TILE_SIZE];
//	// переменная для вычисления элемента подматрицы
//	double sum = 0.0;
//	for (int ia = aBegin, ib = bBegin; ia < aEnd; ia += aStep, ib += bStep)
//	{
//		// загрузка подматриц А и В из глобальной памяти в
//		// разделяемую
//		as[threadIdx.y][threadIdx.x] = A[ia + Acols * threadIdx.y + threadIdx.x];
//		bs[threadIdx.y][threadIdx.x] = B[ib + Bcols * threadIdx.y + threadIdx.x];
//		// синхронизация нитей
//		__syncthreads();
//		// перемножение двух матриц
//		for (int k = 0; k < blockDim.x; k++)
//		{
//			sum += as[threadIdx.y][k] * bs[k][threadIdx.x];
//		}
//		// синхронизация нитей
//		__syncthreads();
//	}
//	// индекс результирующего элемента в глобальной памяти
//	int ind = Bcols * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
//	// запись элемента в глобальную память
//	C[ind] = sum;
//}
int main()
{
	srand(time(NULL));

	ll* a = NULL, * devA = NULL, * b = NULL, * devB = NULL, * resultCPU = NULL, * resultGPU = NULL, * devResult = NULL;
	struct Size aSize, bSize, * resultSize = NULL, * devResultSize = NULL;
	clock_t startCPU, endCPU;
	hipError_t cudaStatus;
	hipEvent_t start, stop;
	hipDeviceProp_t deviceProp;
	const dim3 blockDim(MAX_2D_TREAD_COUNT, MAX_2D_TREAD_COUNT), gridDim((size_t)ceil(FINAL_MATRIX_WIDTH / ((double)blockDim.x)), (size_t)ceil(FINAL_MATRIX_HEIGHT / ((double)blockDim.y)));
	float milliseconds = 0;

	aSize.width = FINAL_MATRIX_HEIGHT;
	aSize.height = FINAL_MATRIX_WIDTH;
	a = new ll[aSize.width * aSize.height];

	bSize.width = FINAL_MATRIX_WIDTH;
	bSize.height = FINAL_MATRIX_HEIGHT;
	b = new ll[bSize.width * bSize.height];

	fillMatrix(a, aSize);
	fillMatrix(b, bSize);

	printMatrix(a, aSize, "A");
	printMatrix(b, bSize, "B");

	resultCPU = new ll[aSize.height * bSize.width];

	resultSize = new Size();
	printf("CPU start calculation\n");
	startCPU = clock();
	(*resultSize) = matrixMult(a, b, resultCPU, aSize, bSize);
	endCPU = clock();
	milliseconds = CALC_TIME_MS(startCPU, endCPU);

	printMatrix(resultCPU, *resultSize, "CPU result");
	printf("CPU time: %f ms\n", milliseconds);

	///////////////////////////////////////GPU/////////////////////////////////////////////////////

	cudaStatus = hipEventCreate(&start);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventCreate(&start) failed!");

	cudaStatus = hipEventCreate(&stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventCreate(&stop) failed!");

	cudaStatus = hipGetDeviceProperties(&deviceProp, 0);
	CHECK_CUDA_ERROR(cudaStatus, "hipGetDeviceProperties failed!");

	printDeviceProperties(deviceProp);
	cudaStatus = hipMalloc(&devA, aSize.width * aSize.height * sizeof(ll));
	CHECK_CUDA_ERROR(cudaStatus, "hipMalloc(&devA failed!");

	cudaStatus = hipMalloc(&devB, bSize.width * bSize.height * sizeof(ll));
	CHECK_CUDA_ERROR(cudaStatus, "hipMalloc(&devB failed!");

	cudaStatus = hipMalloc(&devResult, aSize.height * bSize.width * sizeof(ll));
	CHECK_CUDA_ERROR(cudaStatus, "hipMalloc(&devResult failed!");

	cudaStatus = hipMalloc(&devResultSize, sizeof(Size));
	CHECK_CUDA_ERROR(cudaStatus, "hipMalloc(&devResultSize failed!");

	cudaStatus = hipMemcpy(devA, a, aSize.width * aSize.height * sizeof(ll), hipMemcpyHostToDevice);
	CHECK_CUDA_ERROR(cudaStatus, "hipMemcpy(devA failed!");

	cudaStatus = hipMemcpy(devB, b, bSize.width * bSize.height * sizeof(ll), hipMemcpyHostToDevice);
	CHECK_CUDA_ERROR(cudaStatus, "hipMemcpy(devB failed!");

	printf("GPU start calculation\n");
	cudaStatus = hipEventRecord(start);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventRecord(&start) failed!");

	matrixMultGPUShared << <gridDim, blockDim >> > (devA, devB, devResult, devResultSize, aSize, bSize);

	cudaStatus = hipGetLastError();
	CHECK_CUDA_ERROR(cudaStatus, "hipGetLastError failed!");

	cudaStatus = hipDeviceSynchronize();
	CHECK_CUDA_ERROR(cudaStatus, "hipDeviceSynchronize failed!");

	cudaStatus = hipEventRecord(stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventRecord(&stop) failed!");

	// Ждем завершения всех операций
	cudaStatus = hipEventSynchronize(stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventSynchronize(&stop) failed!");

	cudaStatus = hipEventElapsedTime(&milliseconds, start, stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventElapsedTime failed!");

	cudaStatus = hipMemcpy(resultSize, devResultSize, sizeof(Size), hipMemcpyDeviceToHost);
	CHECK_CUDA_ERROR(cudaStatus, "hipMemcpy(&resultSize failed!");

	resultGPU = new ll[resultSize->width * resultSize->height];

	cudaStatus = hipMemcpy(resultGPU, devResult, resultSize->width * resultSize->height * sizeof(ll), hipMemcpyDeviceToHost);
	CHECK_CUDA_ERROR(cudaStatus, "hipMemcpy(resultGPU failed!");

	printMatrix(resultGPU, *resultSize, "GPU result");
	printf("GPU time: %f ms\n", milliseconds);

Finish:

	DELETE_ARRAY_IF_EXISTS(a);
	DELETE_ARRAY_IF_EXISTS(b);
	DELETE_ARRAY_IF_EXISTS(resultCPU);
	DELETE_ARRAY_IF_EXISTS(resultGPU);
	DELETE_IF_EXISTS(resultSize);

	// Освобождаем ресурсы
	cudaStatus = hipEventDestroy(start);
	PRINT_CUDA_ERROR(cudaStatus, "hipEventDestroy(start failed!");

	cudaStatus = hipEventDestroy(stop);
	PRINT_CUDA_ERROR(cudaStatus, "hipEventDestroy(stop failed!");

	if (devA)
	{
		cudaStatus = hipFree(devA);
		PRINT_CUDA_ERROR(cudaStatus, "hipFree(devA failed!");
	}

	if (devB)
	{
		cudaStatus = hipFree(devB);
		PRINT_CUDA_ERROR(cudaStatus, "hipFree(devB failed!");
	}

	if (devResult)
	{
		cudaStatus = hipFree(devResult);
		PRINT_CUDA_ERROR(cudaStatus, "hipFree(devResult failed!");
	}

	if (devResultSize)
	{
		cudaStatus = hipFree(devResultSize);
		PRINT_CUDA_ERROR(cudaStatus, "hipFree(devResultSize failed!");
	}
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	PRINT_CUDA_ERROR(cudaStatus, "hipDeviceReset failed!");

	system("pause");
	return 0;
}
#endif
