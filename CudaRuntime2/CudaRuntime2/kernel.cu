#include "hip/hip_runtime.h"
﻿#pragma once
#ifndef __INTELLISENSE_

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <cmath>

// размер блока или размер подматрицы
#define ll long long
#define FINAL_MATRIX_HEIGHT 1000.0
#define FINAL_MATRIX_WIDTH 1000.0
#define MAX_2D_TREAD_COUNT 32.0
#define TILE_SIZE (size_t)MAX_2D_TREAD_COUNT
#define CALC_TIME_MS(start, end) (((double)((end) - (start)) * 1000.0) / CLOCKS_PER_SEC)

#define DELETE_IF_EXISTS(ptr) \
    if (ptr) \
    { \
        delete ptr; \
    }

#define DELETE_ARRAY_IF_EXISTS(ptr) \
    if (ptr) \
    { \
        delete[] ptr; \
    }

#define CHECK_CUDA_ERROR(cudaStatus, message) \
    if ((cudaStatus) != hipSuccess) \
    { \
        fprintf(stderr, message); \
		fprintf(stderr, "CUDA error string:  %s\n", hipGetErrorString(cudaStatus)); \
        goto Finish; \
    }

#define PRINT_CUDA_ERROR(cudaStatus, message) \
    if ((cudaStatus) != hipSuccess) \
    { \
        fprintf(stderr, message); \
		fprintf(stderr, "CUDA error string:  %s\n", hipGetErrorString(cudaStatus)); \
    }

struct Size
{
	size_t width = 0;
	size_t height = 0;
};

__host__ void fillMatrix(ll* matrix, const struct Size size)
{
	for (size_t i = 0; i < size.width * size.height; ++i)
	{
		matrix[i] = i;
	}
}

__host__ void printMatrix(ll* matrix, const struct Size size, const char* matrixName)
{
	printf("Matrix %s:\n", matrixName);
	for (size_t i = 0; i < size.height; ++i)
	{
		for (size_t j = 0; j < size.width; ++j)
		{
			printf("%lld ", matrix[size.width * i + j]);
		}
		printf("\n");
	}
}

__host__ struct Size matrixMult(const ll* a, const ll* b, ll** result, struct Size aSize, struct Size bSize)
{
	struct Size resultSize;
	resultSize.width = bSize.width;
	resultSize.height = aSize.height;
	(*result) = new ll[resultSize.width * resultSize.height];

	size_t n = aSize.width;
	for (size_t i = 0; i < resultSize.height; ++i)
	{
		for (size_t j = 0; j < resultSize.width; ++j)
		{
			size_t index = i * resultSize.width + j;
			(*result)[index] = 0;
			for (size_t k = 0; k < n; ++k)
			{
				(*result)[index] += a[i * aSize.width + k] * b[k * bSize.width + j];
			}
		}
	}

	return resultSize;
}

__host__ void printDeviceProperties(const hipDeviceProp_t& deviceProp)
{
	// Основная информация
	printf("\n\nGPU: %s\n", deviceProp.name);
	printf("Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
	printf("Global Memory: %.2f GB\n", deviceProp.totalGlobalMem / 1073741824.0);

	// Блоки и сетка
	printf("Max Threads per Block: %d\n", deviceProp.maxThreadsPerBlock);
	printf("Max Block Dim: (%d, %d, %d)\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
	printf("Max Grid Dim: (%d, %d, %d)\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);

	// Аппаратные характеристики
	printf("Max blocks per multiprocessor: %d\n", deviceProp.maxBlocksPerMultiProcessor);
	printf("Multiprocessors: %d\n", deviceProp.multiProcessorCount);
	printf("Clock Rate: %.2f GHz\n", deviceProp.clockRate * 1e-6f);
	printf("Shared Memory per Block: %zu KB\n", deviceProp.sharedMemPerBlock / 1024);

	// Дополнительно
	printf("Warp Size: %d\n", deviceProp.warpSize);
	printf("Concurrent Kernels: %s\n", deviceProp.concurrentKernels ? "Yes" : "No");
	printf("Integrated GPU: %s\n\n\n", deviceProp.integrated ? "Yes" : "No");
}

__global__ void matrixMultGPU(const ll* a, const ll* b, ll* result, struct Size* resultSize, struct Size aSize, struct Size bSize)
{
	size_t indexX = blockDim.x * blockIdx.x + threadIdx.x;
	size_t indexY = blockDim.y * blockIdx.y + threadIdx.y;

	if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0)
	{
		resultSize->width = bSize.width;
		resultSize->height = aSize.height;
	}

	if (indexX >= bSize.width || indexY >= aSize.height)
	{
		return;
	}

	ll sum = 0ll;
	for (size_t k = 0; k < aSize.width; ++k)
	{
		sum += a[indexY * aSize.width + k] * b[k * bSize.width + indexX];
	}

	size_t index = bSize.width * indexY + indexX;
	result[index] = sum;
}

__global__ void matrixMultGPUShared(const ll* a, const ll* b, ll* result, struct Size* resultSize, struct Size aSize, struct Size bSize)
{
	if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0)
	{
		resultSize->width = bSize.width;
		resultSize->height = aSize.height;
	}

	__shared__ ll tileA[TILE_SIZE][TILE_SIZE];
	__shared__ ll tileB[TILE_SIZE][TILE_SIZE];

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	ll sum = 0;
	for (int tileIdx = 0; tileIdx < (aSize.width + TILE_SIZE - 1) / TILE_SIZE; ++tileIdx)
	{
		// Загрузка tileA
		int aRow = blockIdx.y * TILE_SIZE + threadIdx.y;
		int aCol = tileIdx * TILE_SIZE + threadIdx.x;
		if (aRow < aSize.height && aCol < aSize.width)
		{
			tileA[threadIdx.y][threadIdx.x] = a[aRow * aSize.width + aCol];
		}
		else
		{
			tileA[threadIdx.y][threadIdx.x] = 0;
		}

		// Загрузка tileB  
		int bRow = tileIdx * TILE_SIZE + threadIdx.y;
		int bCol = blockIdx.x * TILE_SIZE + threadIdx.x;
		if (bRow < bSize.height && bCol < bSize.width)
		{
			tileB[threadIdx.y][threadIdx.x] = b[bRow * bSize.width + bCol];
		}
		else
		{
			tileB[threadIdx.y][threadIdx.x] = 0;
		}
		__syncthreads();

		// Вычисление частичной суммы
		for (int k = 0; k < TILE_SIZE; ++k)
		{
			sum += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
		}

		__syncthreads();
	}

	// Запись результата
	if (row < resultSize->height && col < resultSize->width)
	{
		result[row * resultSize->width + col] = sum;
	}
}

__host__ void CPU(ll* a, ll* b, const Size& aSize, const Size& bSize)
{
	clock_t startCPU, endCPU;
	printf("CPU start calculation\n");
	ll* resultCPU = new ll[0];

	startCPU = clock();
	Size resultSize = matrixMult(a, b, &resultCPU, aSize, bSize);
	endCPU = clock();
	float milliseconds = CALC_TIME_MS(startCPU, endCPU);

	//printMatrix(resultCPU, resultSize, "CPU result");
	printf("CPU time: %f ms\n", milliseconds);
	delete[] resultCPU;
}

__host__ void GPU(ll* a, ll* b, const Size& aSize, const Size& bSize)
{
	ll* resultGPU = NULL, * devA = NULL, * devB = NULL, * devResult = NULL, * result = NULL;
	struct Size* resultSize = NULL, * devResultSize = NULL;
	hipError_t cudaStatus;
	hipEvent_t start, stop;
	const dim3 blockDim(MAX_2D_TREAD_COUNT, MAX_2D_TREAD_COUNT), gridDim((size_t)ceil(FINAL_MATRIX_WIDTH / ((double)blockDim.x)), (size_t)ceil(FINAL_MATRIX_HEIGHT / ((double)blockDim.y)));
	float milliseconds = 0;

	cudaStatus = hipEventCreate(&start);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventCreate(&start) failed!");

	cudaStatus = hipEventCreate(&stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventCreate(&stop) failed!");

	cudaStatus = hipMalloc(&devA, aSize.width * aSize.height * sizeof(ll));
	CHECK_CUDA_ERROR(cudaStatus, "hipMalloc(&devA failed!");

	cudaStatus = hipMalloc(&devB, bSize.width * bSize.height * sizeof(ll));
	CHECK_CUDA_ERROR(cudaStatus, "hipMalloc(&devB failed!");

	cudaStatus = hipMalloc(&devResult, aSize.height * bSize.width * sizeof(ll));
	CHECK_CUDA_ERROR(cudaStatus, "hipMalloc(&devResult failed!");

	cudaStatus = hipMalloc(&devResultSize, sizeof(Size));
	CHECK_CUDA_ERROR(cudaStatus, "hipMalloc(&devResultSize failed!");

	cudaStatus = hipMemcpy(devA, a, aSize.width * aSize.height * sizeof(ll), hipMemcpyHostToDevice);
	CHECK_CUDA_ERROR(cudaStatus, "hipMemcpy(devA failed!");

	cudaStatus = hipMemcpy(devB, b, bSize.width * bSize.height * sizeof(ll), hipMemcpyHostToDevice);
	CHECK_CUDA_ERROR(cudaStatus, "hipMemcpy(devB failed!");

	printf("GPU start calculation\n");
	cudaStatus = hipEventRecord(start);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventRecord(&start) failed!");

	matrixMultGPU<<<gridDim, blockDim>>>(devA, devB, devResult, devResultSize, aSize, bSize);

	cudaStatus = hipGetLastError();
	CHECK_CUDA_ERROR(cudaStatus, "hipGetLastError failed!");

	cudaStatus = hipDeviceSynchronize();
	CHECK_CUDA_ERROR(cudaStatus, "hipDeviceSynchronize failed!");

	cudaStatus = hipEventRecord(stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventRecord(&stop) failed!");

	// Ждем завершения всех операций
	cudaStatus = hipEventSynchronize(stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventSynchronize(&stop) failed!");

	cudaStatus = hipEventElapsedTime(&milliseconds, start, stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventElapsedTime failed!");

	resultSize = new Size();
	cudaStatus = hipMemcpy(resultSize, devResultSize, sizeof(Size), hipMemcpyDeviceToHost);
	CHECK_CUDA_ERROR(cudaStatus, "hipMemcpy(&resultSize failed!");

	resultGPU = new ll[resultSize->width * resultSize->height];

	cudaStatus = hipMemcpy(resultGPU, devResult, resultSize->width * resultSize->height * sizeof(ll), hipMemcpyDeviceToHost);
	CHECK_CUDA_ERROR(cudaStatus, "hipMemcpy(resultGPU failed!");

	//printMatrix(resultGPU, *resultSize, "\nGPU result");
	printf("GPU time: %f ms\n", milliseconds);

Finish:
	DELETE_ARRAY_IF_EXISTS(resultGPU);
	DELETE_IF_EXISTS(resultSize);

	// Освобождаем ресурсы
	cudaStatus = hipEventDestroy(start);
	PRINT_CUDA_ERROR(cudaStatus, "hipEventDestroy(start failed!");

	cudaStatus = hipEventDestroy(stop);
	PRINT_CUDA_ERROR(cudaStatus, "hipEventDestroy(stop failed!");

	if (devA)
	{
		cudaStatus = hipFree(devA);
		PRINT_CUDA_ERROR(cudaStatus, "hipFree(devA failed!");
	}

	if (devB)
	{
		cudaStatus = hipFree(devB);
		PRINT_CUDA_ERROR(cudaStatus, "hipFree(devB failed!");
	}

	if (devResult)
	{
		cudaStatus = hipFree(devResult);
		PRINT_CUDA_ERROR(cudaStatus, "hipFree(devResult failed!");
	}

	if (devResultSize)
	{
		cudaStatus = hipFree(devResultSize);
		PRINT_CUDA_ERROR(cudaStatus, "hipFree(devResultSize failed!");
	}
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	PRINT_CUDA_ERROR(cudaStatus, "hipDeviceReset failed!");
}

__host__ void GPUShared(ll* a, ll* b, const Size& aSize, const Size& bSize)
{
	ll* resultGPU = NULL, * devA = NULL, * devB = NULL, * devResult = NULL, * result = NULL;
	struct Size* resultSize = NULL, * devResultSize = NULL;
	hipError_t cudaStatus;
	hipEvent_t start, stop;
	const dim3 blockDim(MAX_2D_TREAD_COUNT, MAX_2D_TREAD_COUNT), gridDim((size_t)ceil(FINAL_MATRIX_WIDTH / ((double)blockDim.x)), (size_t)ceil(FINAL_MATRIX_HEIGHT / ((double)blockDim.y)));
	float milliseconds = 0;

	cudaStatus = hipEventCreate(&start);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventCreate(&start) failed!");

	cudaStatus = hipEventCreate(&stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventCreate(&stop) failed!");

	cudaStatus = hipMalloc(&devA, aSize.width * aSize.height * sizeof(ll));
	CHECK_CUDA_ERROR(cudaStatus, "hipMalloc(&devA failed!");

	cudaStatus = hipMalloc(&devB, bSize.width * bSize.height * sizeof(ll));
	CHECK_CUDA_ERROR(cudaStatus, "hipMalloc(&devB failed!");

	cudaStatus = hipMalloc(&devResult, aSize.height * bSize.width * sizeof(ll));
	CHECK_CUDA_ERROR(cudaStatus, "hipMalloc(&devResult failed!");

	cudaStatus = hipMalloc(&devResultSize, sizeof(Size));
	CHECK_CUDA_ERROR(cudaStatus, "hipMalloc(&devResultSize failed!");

	cudaStatus = hipMemcpy(devA, a, aSize.width * aSize.height * sizeof(ll), hipMemcpyHostToDevice);
	CHECK_CUDA_ERROR(cudaStatus, "hipMemcpy(devA failed!");

	cudaStatus = hipMemcpy(devB, b, bSize.width * bSize.height * sizeof(ll), hipMemcpyHostToDevice);
	CHECK_CUDA_ERROR(cudaStatus, "hipMemcpy(devB failed!");

	printf("GPU start calculation\n");
	cudaStatus = hipEventRecord(start);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventRecord(&start) failed!");

	matrixMultGPUShared<<<gridDim, blockDim>>>(devA, devB, devResult, devResultSize, aSize, bSize);

	cudaStatus = hipGetLastError();
	CHECK_CUDA_ERROR(cudaStatus, "hipGetLastError failed!");

	cudaStatus = hipDeviceSynchronize();
	CHECK_CUDA_ERROR(cudaStatus, "hipDeviceSynchronize failed!");

	cudaStatus = hipEventRecord(stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventRecord(&stop) failed!");

	// Ждем завершения всех операций
	cudaStatus = hipEventSynchronize(stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventSynchronize(&stop) failed!");

	cudaStatus = hipEventElapsedTime(&milliseconds, start, stop);
	CHECK_CUDA_ERROR(cudaStatus, "hipEventElapsedTime failed!");

	resultSize = new Size();
	cudaStatus = hipMemcpy(resultSize, devResultSize, sizeof(Size), hipMemcpyDeviceToHost);
	CHECK_CUDA_ERROR(cudaStatus, "hipMemcpy(&resultSize failed!");

	resultGPU = new ll[resultSize->width * resultSize->height];

	cudaStatus = hipMemcpy(resultGPU, devResult, resultSize->width * resultSize->height * sizeof(ll), hipMemcpyDeviceToHost);
	CHECK_CUDA_ERROR(cudaStatus, "hipMemcpy(resultGPU failed!");

	//printMatrix(resultGPU, *resultSize, "\nGPUShared result");
	printf("GPUShared time: %f ms\n", milliseconds);

Finish:
	DELETE_ARRAY_IF_EXISTS(resultGPU);
	DELETE_IF_EXISTS(resultSize);

	// Освобождаем ресурсы
	cudaStatus = hipEventDestroy(start);
	PRINT_CUDA_ERROR(cudaStatus, "hipEventDestroy(start failed!");

	cudaStatus = hipEventDestroy(stop);
	PRINT_CUDA_ERROR(cudaStatus, "hipEventDestroy(stop failed!");

	if (devA)
	{
		cudaStatus = hipFree(devA);
		PRINT_CUDA_ERROR(cudaStatus, "hipFree(devA failed!");
	}

	if (devB)
	{
		cudaStatus = hipFree(devB);
		PRINT_CUDA_ERROR(cudaStatus, "hipFree(devB failed!");
	}

	if (devResult)
	{
		cudaStatus = hipFree(devResult);
		PRINT_CUDA_ERROR(cudaStatus, "hipFree(devResult failed!");
	}

	if (devResultSize)
	{
		cudaStatus = hipFree(devResultSize);
		PRINT_CUDA_ERROR(cudaStatus, "hipFree(devResultSize failed!");
	}
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	PRINT_CUDA_ERROR(cudaStatus, "hipDeviceReset failed!");
}

long main()
{
	srand(time(NULL));
	ll* a = NULL, * b = NULL;
	struct Size aSize, bSize;

	aSize.width = FINAL_MATRIX_HEIGHT;
	aSize.height = FINAL_MATRIX_WIDTH;
	a = new ll[aSize.width * aSize.height];

	bSize.width = FINAL_MATRIX_WIDTH;
	bSize.height = FINAL_MATRIX_HEIGHT;
	b = new ll[bSize.width * bSize.height];

	fillMatrix(a, aSize);
	fillMatrix(b, bSize);

	/*printMatrix(a, aSize, "A");
	printMatrix(b, bSize, "B");*/

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	printDeviceProperties(deviceProp);

	CPU(a, b, aSize, bSize);
	GPU(a, b, aSize, bSize);
	GPUShared(a, b, aSize, bSize);

	delete[] a;
	delete[] b;
	system("pause");
	return 0;
}
#endif